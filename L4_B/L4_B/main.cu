﻿#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#include <vector>
#include <fstream>
#include <string>
#include <sstream>
#include <iostream>
#pragma comment(lib, "cudart.lib")

using namespace std;
struct DataSet 
{
	int i;
	double d;
	int length;
	char s[200];

	__host__ __device__
	DataSet()
	{
		i = 0;
		d = 0;
		length = 0;
	}
	__host__ __device__
	DataSet(int j)
	{
		i = 0;
		d = 0;
		length = 0;		
	}
};
 __device__
DataSet operator+(const DataSet& a, const DataSet& b)
{
	DataSet r;
	r.d = a.d + b.d;
	r.i = a.i + b.i;
	memcpy(r.s, a.s, a.length);
	memcpy(r.s + a.length, b.s, b.length);
	r.length = a.length + b.length;
	r.s[r.length] = 0;
	return r;
}



vector<vector<DataSet>> readFile(const char* filename);
int main(int argc, char** argv)
{
	vector<vector<DataSet>> data = readFile("DapseviciusA_L4.txt");
	vector<DataSet> flatData;
	// Raktų masyvas nurodantis elementų indeksus
	vector<int> keys;
	// Duomenų masyvas perdaromas į vienmatį masyvą
	int keys_count = 0;
	for(int i = 0; i < data.size(); i++)
	{
		int j = 0;
		for(; j < data[i].size(); j++)
		{
			keys.emplace_back(j);
			flatData.emplace_back(data[i][j]);
		}
		if(keys_count < j)
		{
			keys_count = j;
		}
	}
	
    thrust::equal_to<int> binary_pred;
    thrust::plus<DataSet> binary_op;
	thrust::sort_by_key(keys.data(), keys.data() + keys.size(), flatData.data());

	thrust::host_vector<int> host_keys = keys;
	thrust::device_vector<int> device_keys = host_keys;

	thrust::host_vector<DataSet> host_values = flatData;
	thrust::device_vector<DataSet> device_values = host_values;

	thrust::device_vector<DataSet> output_values;
	output_values.reserve(flatData.size());
	thrust::device_vector<int> output_keys;
	output_keys.reserve(flatData.size());

	try
	{
		//p = thrust::reduce_by_key(keys.data(), keys.data() + keys.size(), flatData.data(), result_keys, result_values);
		auto p = thrust::reduce_by_key(device_keys.begin(), device_keys.end(), device_values.begin(), output_keys.begin(), output_values.begin(), binary_pred, binary_op);
	}
	catch(thrust::system_error &e)
	{
	// output an error message and exit
		std::cerr << "Error " << e.what() << std::endl;
		exit(-1);
	}

	thrust::host_vector<int> result_keys = output_keys;
	
	for(int i = 0; i < keys_count; i++)
	{
		cout << "Sudėjus elementus kurių indeksas yra " << output_keys[i] << " gauta:" << endl;
		DataSet d = output_values[i];
		printf("%3.3f, %3d, %s\n\n", d.d, d.i, d.s);
	}
	system("pause");
	return 0;
}

vector<vector<DataSet>> readFile(const char* filename)
{
	cout << "Pradiniai duomenys:" << endl;
	vector<vector<DataSet>> result;
	ifstream file(filename);
	if(file.good())
	{
		int count;
		file >> count;
		cout << "Masyvu kiekis:	" << count << endl;
		for(int i = 0; i < count; i++)
		{			
			vector<DataSet> temp;
			int itemCount;
			file >> itemCount;
			cout << "Masyvo elementu skaicius: " << itemCount << endl;
			cout << "Skaicius	sv. skaicius	eilute" << endl;
			for(int j = 0; j < itemCount; j++)
			{
				string word;
				DataSet current;
				file >> current.i >> current.d;
				getline(file, word);
				int length = word.length() > 20 ? 20 : word.length();
				memcpy(current.s, word.c_str()+1, length);
				current.s[19] = 0;
				current.length = length-1;
				temp.emplace_back(current);
				cout << current.d << "		" << current.i << "		" << current.s << endl;
			}
			cout << endl;
			result.emplace_back(temp);
		}
	}
	else
	{
		cout << "Negeras failas" << endl;
		system("pause");
		exit(-1);
	}
	return result;
}